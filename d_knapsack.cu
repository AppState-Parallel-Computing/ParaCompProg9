#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "helpers.h"
#include "d_knapsack.h"

//prototypes for kernels in this file
__global__ 
void d_knapsackNaiveKernel(int * d_best, int * d_weight, int * d_values, int numObjs,
                           int capacity);

__global__ 
void d_knapsackOptKernel(int * d_best, int * d_weight, int * d_values, int numObjs,
                           int capacity);

/*  d_knapsack
    This function prepares and invokes a kernel to solve the 0-1 knapsack problem
    on the GPU. The input to the knapsack problem is a set of objects and a 
    knapsack capacity.  Each object has a weight and a value. The solution chooses a subset 
    of the objects that maximums the overall value while not exceeding the capacity.
    Inputs:
    result - points to an array to hold the knapsack result
    weights - points to an array that holds the weights of the objects
    values - points to an array that holds the values of the objects
    numObjs - number of objects (size of values and weights arrays)
    capacity - the capacity of the knapsack
    blkDim - the number of threads in the block of threads used to solve the problem
    which - indicates which kernel to use to solve the problem (NAIVE, OPT)
*/
float d_knapsack(int * result, int * weights, int * values, int numObjs, 
                 int capacity, int blkDim, int which)
{
    int * d_best, * d_weights, * d_values;  //pointers to arrays for GPU
   
    //CUERR is a macro in helpers.h that checks for a Cuda error 
    //Begin the timing (macro in helpers.h) 
    TIMERSTART(gpuTime)

    //Allocate space in GPU memory for weights array 
    hipMalloc((void **)&d_weights, sizeof(int) * numObjs);             CUERR
    //Copy weights from CPU memory to GPU memory
    hipMemcpy(d_weights, weights, sizeof(int) * numObjs, H2D);         CUERR

    //Allocate space in GPU memory for values array 
    hipMalloc((void **)&d_values, sizeof(int) * numObjs);              CUERR
    //Copy values from CPU memory to GPU memory
    hipMemcpy(d_values, values, sizeof(int) * numObjs, H2D);           CUERR

    //Launch the appropriate kernel
    if (which == NAIVE)
    {
        //Allocate space in GPU memory for best matrix
        int bestSz = (numObjs + 1) * (capacity + 1);
        hipMalloc((void **)&d_best, sizeof(int) * bestSz);             CUERR
        //set the best matrix to 0
        hipMemset((void *)d_best, 0, bestSz * sizeof(int));            CUERR
        //define the block and the grid and launch the naive kernel
        dim3 block(blkDim, 1, 1);
        dim3 grid(1, 1, 1);
        d_knapsackNaiveKernel<<<grid, block>>>(d_best, d_weights, d_values,
                                               numObjs, capacity);     CUERR
        //copy last row of d_best array into result
        hipMemcpy(result, 
                   &d_best[numObjs * (capacity + 1)], sizeof(int) * (capacity + 1),
                   D2H);                                               CUERR
    } else if (which == OPT)
    {
        //TO DO
        //Provide the code that is missing to execute the optimized kernel

    }
    //free dynamically  allocated memory
    hipFree(d_best);                                                 CUERR
    hipFree(d_values);                                               CUERR
    hipFree(d_weights);                                              CUERR

    //stop the timer
    TIMERSTOP(gpuTime)
    return TIMEELAPSED(gpuTime)
}

/*  
    d_knapsackNaiveKernel
    This kernel solves the knapsack problem using a naive kernel.
    Inputs:
    best - pointer to the array in which the result is stored
    weights - points to an array that holds the weights of the objects
    values - points to an array that holds the values of the objects
    numObjs - number of objects (size of values and weights arrays)
    capacity - the capacity of the knapsack
*/

__global__
void d_knapsackNaiveKernel(int * best, int * weights, int * values, 
                           int numObjs, int capacity)
{
    //TO DO

    //You should base this implementation on the CPU version in h_knapsack.cu, 
    //but the best array needs to be allocated before the kernel launch.

    //All threads of a block will cooperate in producing one row (i) of results.
    //Block synchronization is needed so the threads in a block won't continue
    //onto next row until all threads are finished with the current row.

    //The elements of a row are distributed among the threads in a block in
    //a cyclic manner.

    //The thread identifier is used by the thread to choose the first element
    //within the row that it is responsible for.  For example, for i equal 
    //to 0, thread 0 will write to best[0], best[blockDim.x], best[2*blockDim.x], etc.

}

/*  
    d_knapsackOptKernel
    This kernel solves the knapsack problem using an optimized kernel.
    Inputs:
    best - pointer to the array in which the result is stored
    weights - points to an array that holds the weights of the objects
    values - points to an array that holds the values of the objects
    numObjs - number of objects (size of values and weights arrays)
    capacity - the capacity of the knapsack
*/
__global__ 
void d_knapsackOptKernel(int * best, int * weights, int * values, int numObjs,
                         int capacity)
{
    //TO DO

    //For this one, start with the naive kernel code and improve it.
    //Specifically, reduce the number of accesses to global memory where you can.
    //Instead those accesses should access registers or shared memory.
    //This requires just a very simple modification of the code.

    //Second, use less global memory: O(capacity) instead of O(capacity * numObjs)
}      

